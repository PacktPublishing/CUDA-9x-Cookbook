#include "hip/hip_runtime.h"
#include <iostream>
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h> // Needed or __global__ == unrecognised.
#include "" // Variable identifiers.
#include <memory>
#include <array>
#include "CudaContainer.h"

const int SIZE = 256;
const int THREADS_PER_BLOCK = 4;
const int NO_OF_BLOCKS = SIZE / THREADS_PER_BLOCK;

__global__ void device_add(int* a, int* b, int* c);
void fill_array(const std::shared_ptr<std::array<int, SIZE>>& out);

int main()
{
	std::cout << "Hello" << std::endl;

	// Host memory allocation.
	std::shared_ptr<std::array<int, SIZE>> a = std::make_shared<std::array<int, SIZE>>();
	std::shared_ptr<std::array<int, SIZE>> b = std::make_shared<std::array<int, SIZE>>();
	std::shared_ptr<std::array<int, SIZE>> c = std::make_shared<std::array<int, SIZE>>();
	// Device memory allocation.
	std::shared_ptr<CudaContainer<int>> d_a = std::make_shared<CudaContainer<int>>(SIZE);
	std::shared_ptr<CudaContainer<int>> d_b = std::make_shared<CudaContainer<int>>(SIZE);
	std::shared_ptr<CudaContainer<int>> d_c = std::make_shared<CudaContainer<int>>(SIZE);

	fill_array(a);
	fill_array(b);

	hipMemcpy(d_a->data, a.get(), SIZE * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_b->data, b.get(), SIZE * sizeof(int), hipMemcpyHostToDevice);

	device_add << <NO_OF_BLOCKS, THREADS_PER_BLOCK >> > (d_a->data, d_b->data, d_c->data);

	hipDeviceSynchronize();

	hipMemcpy(c.get(), d_c->data, SIZE * sizeof(int), hipMemcpyDeviceToHost);

	// No need to manually call free or hipFree.
	// Since hipFree is in the destructor of ~CudaContainer.
	// Which is wrapped in a shared_ptr.

	for (int i = 0; i < SIZE; i++)
		std::cout << (*c)[i] << std::endl;

	return 0;
}

template <class T>
CudaContainer<T>::CudaContainer(int size)
{
	this->size = size;
	hipMalloc(&data, size * sizeof(T));
}

template <class T>
CudaContainer<T>::~CudaContainer()
{
	hipFree(data);
}

__global__ void device_add(int* a, int* b, int* c)
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	printf("Setting index %d to %d + %d\n", index, a[index], b[index]);
	c[index] = a[index] + b[index];
}

void fill_array(const std::shared_ptr<std::array<int, SIZE>>& out)
{
	for (int i = 0; i < SIZE; i++)
		(*out)[i] = i;
}